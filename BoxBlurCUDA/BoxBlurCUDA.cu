#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include <stdlib.h>
#include "lodepng.h"

__global__ void boxBlur(int w, int h, unsigned char *gpu_in, unsigned char *gpu_out){

    int i = blockIdx.x;
    int j = threadIdx.x;

    if(i == 0 && j == 0){
		int a = w * i * 4 + j * 4;
		int b = w * i * 4 + (j + 1) * 4;
		int c = w *(i + 1)*4 + j*4;
		int d = w *(i + 1)*4 + (j + 1)*4;
		gpu_out[a + 0] = (gpu_in[a+0] + gpu_in[b+0] + gpu_in[c+0] + gpu_in[d+0])/4;
		gpu_out[a + 1] = (gpu_in[a+1] + gpu_in[b+1] + gpu_in[c+1] + gpu_in[d+1])/4;
		gpu_out[a + 2] = (gpu_in[a+2] + gpu_in[b+2] + gpu_in[c+2] + gpu_in[d+2])/4;
    }else if(i == 0 && j > 0 && j != (w - 1)){
        int a = w * i * 4 + j * 4;
        int b = w * i * 4 + (j - 1) * 4;
        int c = w * i * 4 + (j + 1) * 4;
        int d = w *(i+1)*4 + (j - 1)*4;
        int e = w *(i+1)*4 + j*4;
        int f = w *(i+1)*4 + (j + 1)*4;
        gpu_out[a + 0] = (gpu_in[b+0] + gpu_in[a+0] + gpu_in[c+0] + gpu_in[d+0] + gpu_in[e+0] + gpu_in[f+0])/6;
        gpu_out[a + 1] = (gpu_in[b + 1]+gpu_in[a + 1]+gpu_in[c + 1]+gpu_in[d + 1]+gpu_in[e + 1]+gpu_in[f + 1])/6;		
        gpu_out[a + 2] = (gpu_in[b + 2]+gpu_in[a + 2]+gpu_in[c + 2]+gpu_in[d + 2]+gpu_in[e + 2]+gpu_in[f + 2])/6;		
	}else if(i == 0 && j == (w-1)){
        int a = w * i * 4 + (j - 1) * 4;
        int b = w * i * 4 + j * 4;
        int c = w *(i + 1)*4 + (j - 1)*4;
        int d = w *(i + 1)*4 + j*4;
        gpu_out[b + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0])/4;
        gpu_out[b + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1])/4;
        gpu_out[b + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2])/4;
    }else if(i > 0 && j == 0 && i != (h-1)){
        int a = w * (i-1) * 4 + j * 4;
        int b = w * (i-1) * 4 + (j+1) * 4;
        int c = w * i * 4 + j * 4;
        int d = w * i * 4 + (j+1) * 4;
        int e = w * (i+1) * 4 + j * 4;
        int f = w * (i+1) * 4 + (j+1) * 4;
        gpu_out[c + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0]+gpu_in[e + 0]+gpu_in[f + 0])/6;
        gpu_out[c + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1]+gpu_in[e + 1]+gpu_in[f + 1])/6;
        gpu_out[c + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2]+gpu_in[e + 2]+gpu_in[f + 2])/6;
    }else if(i > 0 && j == (w-1) && i != (h-1)){
        int a = w * (i-1) * 4 + (j-1) * 4;
        int b = w * (i-1) * 4 + j * 4;
        int c = w * i * 4 + (j-1) * 4;
        int d = w * i * 4 + j * 4;
        int e = w * (i+1) * 4 + (j-1) * 4;
        int f = w * (i+1) * 4 + j * 4;
        gpu_out[d + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0]+gpu_in[e + 0]+gpu_in[f + 0])/6;
        gpu_out[d + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1]+gpu_in[e + 1]+gpu_in[f + 1])/6;
        gpu_out[d + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2]+gpu_in[e + 2]+gpu_in[f + 2])/6;	
    }else if(i > 0 && j == (w-1) && i != (h-1)){
        int a = w * (i-1) * 4 + (j-1) * 4;
        int b = w * (i-1) * 4 + j * 4;
        int c = w * i * 4 + (j-1) * 4;
        int d = w * i * 4 + j * 4;
        int e = w * (i+1) * 4 + (j-1) * 4;
        int f = w * (i+1) * 4 + j * 4;
        gpu_out[d + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0]+gpu_in[e + 0]+gpu_in[f + 0])/6;
        gpu_out[d + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1]+gpu_in[e + 1]+gpu_in[f + 1])/6;
        gpu_out[d + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2]+gpu_in[e + 2]+gpu_in[f + 2])/6;
    }else if(i == (h-1) && j == 0){
        int a = w * (i-1) * 4 + j * 4;
        int b = w * (i-1) * 4 + (j+1) * 4;
        int c = w * i * 4 + j * 4;
        int d = w * i * 4 + (j+1) * 4;
        gpu_out[c + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0])/4;
        gpu_out[c + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1])/4;
        gpu_out[c + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2])/4;	
    }else if(i == (h-1) && j > 0 && j != (w-1)){
        int a = w * (i-1) * 4 + (j-1) * 4;
        int b = w * (i-1) * 4 + j * 4;
        int c = w * (i-1) * 4 + (j+1) * 4;
        int d = w * i * 4 + (j-1) * 4;
        int e = w * i * 4 + j * 4;
        int f = w * i * 4 + (j+1) * 4;
        gpu_out[e + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0]+gpu_in[e + 0]+gpu_in[f + 0])/6;
        gpu_out[e + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1]+gpu_in[e + 1]+gpu_in[f + 1])/6;
        gpu_out[e + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2]+gpu_in[e + 2]+gpu_in[f + 2])/6;
    }else if (i == (h-1) && j == (w - 1)){
        int a = w * (i-1) * 4 + (j-1) * 4;
        int b = w * (i-1) * 4 + j * 4 ;
        int c = w * i * 4 + (j-1) * 4;
        int d = w * i * 4 + j * 4;
        gpu_out[d + 0] = (gpu_in[a + 0]+gpu_in[b + 0]+gpu_in[c + 0]+gpu_in[d + 0])/4;
        gpu_out[d + 1] = (gpu_in[a + 1]+gpu_in[b + 1]+gpu_in[c + 1]+gpu_in[d + 1])/4;
        gpu_out[d + 2] = (gpu_in[a + 2]+gpu_in[b + 2]+gpu_in[c + 2]+gpu_in[d + 2])/4;	
    }else{
        int a = w * (i-1) * 4 + (j-1) * 4;
        int b = w * (i-1) * 4 + j * 4;
        int c = w * (i-1) * 4 + (j+1) * 4;
        int d = w * i * 4 + (j-1) * 4;
        int e = w * i * 4 + j * 4;
        int f = w * i * 4 + (j+1) * 4;
        int g = w * (i+1) * 4 + (j-1) * 4;
        int h = w * (i+1) * 4 + j * 4;
        int k = w * (i+1) * 4 + (j+1) * 4;
        gpu_out[e + 0] = (gpu_in[a+0]+gpu_in[b+0]+gpu_in[c+0]+gpu_in[d+0]+gpu_in[e+0]+gpu_in[f+0]+gpu_in[g+0]+gpu_in[h+0]+gpu_in[k+0])/9;
        gpu_out[e+1] = (gpu_in[a+1]+gpu_in[b+1]+gpu_in[c+1]+gpu_in[d+1]+gpu_in[e+1]+gpu_in[f+1]+gpu_in[g+1]+gpu_in[h+1]+gpu_in[k+1])/9;
        gpu_out[e+2] = (gpu_in[a+2]+gpu_in[b+2]+gpu_in[c+2]+gpu_in[d+2]+gpu_in[e+2]+gpu_in[f+2]+gpu_in[g+2]+gpu_in[h+2]+gpu_in[k+2])/9;
	}	
	gpu_out[w * i * 4 + j * 4 + 3] = gpu_in[w * i * 4 + j * 4 + 3];	

}

int main(int argc, char **argv){
    if(argc < 2){
		printf("Please provide png image as an argument. \n");
		exit(1);
	}

    unsigned int error;
    unsigned int encError;
    unsigned char *image;
    unsigned int width;
    unsigned int height;
    char *filename = argv[1];
    const char *newImage = "bluredImage.png";

    error = lodepng_decode32_file(&image, &width, &height, filename);
    if(error){
        printf("Decoding error %u: %s\n", error, lodepng_error_text(error));
        exit(1);
    }

    const int ARRAY_SIZE = width * height * 4;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(unsigned char);
    unsigned char cpu_in[ARRAY_SIZE];
    unsigned char cpu_out[ARRAY_SIZE];

    for (int i = 0; i < ARRAY_SIZE; i++) {
        cpu_in[i] = image[i];
    }

    unsigned char *gpu_in;
    unsigned char *gpu_out;
    hipMalloc((void**) &gpu_in, ARRAY_BYTES);
    hipMalloc((void**) &gpu_out, ARRAY_BYTES);

    hipMemcpy(gpu_in, cpu_in, ARRAY_BYTES, hipMemcpyHostToDevice);
    boxBlur<<< height, width >>>(width, height, gpu_in, gpu_out);
    hipMemcpy(cpu_out, gpu_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    encError = lodepng_encode32_file(newImage, cpu_out, width, height);
    if(encError){
        printf("Encoding error %u: %s\n", error, lodepng_error_text(encError));
        exit(1);
    } else{
        printf("\nOutput file name : %s \n\n", newImage);
    }

    free(image);
    hipFree(gpu_in);
    hipFree(gpu_out);
    return 0;
}
