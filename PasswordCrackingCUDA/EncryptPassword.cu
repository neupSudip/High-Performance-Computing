#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__device__ char* CudaCrypt(char* text){

	char * prod_password = (char *) malloc(sizeof(char) * 11);
 
	prod_password[0] = text[0] + 2;
	prod_password[1] = text[0] - 2;
	prod_password[2] = text[0] + 1;
	prod_password[3] = text[1] + 3;
	prod_password[4] = text[1] - 3;
	prod_password[5] = text[1] - 1;
	prod_password[6] = text[2] + 2;
	prod_password[7] = text[2] - 2;
	prod_password[8] = text[3] + 4;
	prod_password[9] = text[3] - 4;
	prod_password[10] = '\0';

	for(int i =0; i<10; i++){
		if(i >= 0 && i < 6){ 
			if(prod_password[i] > 90){
				prod_password[i] = (prod_password[i] - 90) + 65;
			}else if(prod_password[i] < 65){
				prod_password[i] = (65 - prod_password[i]) + 65;
			}
		}else{ 
			if(prod_password[i] > 57){
				prod_password[i] = (prod_password[i] - 57) + 48;
			}else if(prod_password[i] < 48){
				prod_password[i] = (48 - prod_password[i]) + 48;
			}
		}
	}
	return prod_password;
}

__global__ void crack(char *text){

    printf("\nText      : %s\nEncrypted : %s\n\n", text, CudaCrypt(text));
}

int main(int argc, char ** argv){

	if(argc < 2){
		printf("Please provide text (e.g AA99) as an argument. \n");
		exit(1);
	}
    char *cpu_text = argv[1];
    char *gpu_text;

    hipMalloc( (void**) &gpu_text, sizeof(char) * 4); 
    hipMemcpy(gpu_text, cpu_text, sizeof(char) * 4, hipMemcpyHostToDevice);

    crack<<< 1, 1>>>(gpu_text);
    hipDeviceSynchronize();

	hipFree(gpu_text);
    return 0;	
}
