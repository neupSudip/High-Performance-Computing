#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__device__ char* CudaCrypt(char* rawPassword){
	char * newPassword = (char *) malloc(sizeof(char) * 11);
 
	newPassword[0] = rawPassword[0] + 2;
	newPassword[1] = rawPassword[0] - 2;
	newPassword[2] = rawPassword[0] + 1;
	newPassword[3] = rawPassword[1] + 3;
	newPassword[4] = rawPassword[1] - 3;
	newPassword[5] = rawPassword[1] - 1;
	newPassword[6] = rawPassword[2] + 2;
	newPassword[7] = rawPassword[2] - 2;
	newPassword[8] = rawPassword[3] + 4;
	newPassword[9] = rawPassword[3] - 4;
	newPassword[10] = '\0';

	for(int i = 0; i < 10; i++){
		if(i >= 0 && i < 6){ 
			if(newPassword[i] > 90){
				newPassword[i] = (newPassword[i] - 90) + 65;
			}else if(newPassword[i] < 65){
				newPassword[i] = (65 - newPassword[i]) + 65;
			}
		}else{ 
			if(newPassword[i] > 57){
				newPassword[i] = (newPassword[i] - 57) + 48;
			}else if(newPassword[i] < 48){
				newPassword[i] = (48 - newPassword[i]) + 48;
			}
		}
	}
	return newPassword;
}

__global__ void crack(char * alphabet, char * numbers, char *text, char *in_hash){
    char genRawPass[4];

    genRawPass[0] = alphabet[blockIdx.x];
    genRawPass[1] = alphabet[blockIdx.y];
    genRawPass[2] = numbers[threadIdx.x];
    genRawPass[3] = numbers[threadIdx.y];

	char *prod_hash = CudaCrypt(genRawPass);

	int counter = 0;
	for(int i = 0; i < 10; i++){
		if (prod_hash[i] == in_hash[i]){
			counter++;
		}
	}

	if(counter == 10){
		for (int j = 0; j < 4; j++){
			text[j] = genRawPass[j];
		}
	}
}

int main(int argc, char **argv){

	if(argc < 2){
		printf("Please provide hash (e.g AAAAAA9999) as an argument. \n");
		exit(1);
	}

	char *cpu_text;
	char *gpu_text;

    char *cpuRawPas = argv[1];
    char *gpuRawPass;

    char cpuAlphabet[26] = {'A','B','C','D','E','F','G','H','I','J','K','L','M','N','O','P','Q','R','S','T','U','V','W','X','Y','Z'};
	char *gpuAlphabet;

    char cpuNumbers[26] = {'0','1','2','3','4','5','6','7','8','9'};
	char *gpuNumbers;

	cpu_text = (char *)malloc(4 * sizeof(char));
	
	hipMalloc( (void**) &gpu_text, sizeof(char) * 4); 
	hipMemcpy(gpu_text, cpu_text, sizeof(char) * 4, hipMemcpyHostToDevice);

	hipMalloc( (void**) &gpuRawPass, sizeof(char) * 10); 
    hipMemcpy(gpuRawPass, cpuRawPas, sizeof(char) * 10, hipMemcpyHostToDevice);

    hipMalloc( (void**) &gpuAlphabet, sizeof(char) * 26); 
    hipMemcpy(gpuAlphabet, cpuAlphabet, sizeof(char) * 26, hipMemcpyHostToDevice);

    hipMalloc( (void**) &gpuNumbers, sizeof(char) * 10); 
    hipMemcpy(gpuNumbers, cpuNumbers, sizeof(char) * 10, hipMemcpyHostToDevice);

    crack<<< dim3(26,26,1), dim3(10,10,1) >>>( gpuAlphabet, gpuNumbers ,gpu_text, gpuRawPass);
    hipDeviceSynchronize();

	hipMemcpy(cpu_text, gpu_text, sizeof(char) * 4, hipMemcpyDeviceToHost);

    printf("\nText      : %s\nEncrypted : %s\n\n", cpu_text, cpuRawPas);

	hipFree(gpu_text);
	hipFree(gpuRawPass);
	hipFree(gpuAlphabet);
	hipFree(gpuNumbers);
    return 0;
}

